#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <png.h>
#include <hip/hip_runtime.h>
#include <math.h>

// ------------------ libpng I/O ------------------
unsigned char* readPNG(const char* filename, int* width, int* height, int* channels) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Error opening file %s\n", filename);
        return NULL;
    }

    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) {
        fclose(fp);
        return NULL;
    }
    png_infop info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        fclose(fp);
        return NULL;
    }
    if (setjmp(png_jmpbuf(png_ptr))) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        fclose(fp);
        return NULL;
    }

    png_init_io(png_ptr, fp);
    png_read_info(png_ptr, info_ptr);
    *width    = png_get_image_width(png_ptr, info_ptr);
    *height   = png_get_image_height(png_ptr, info_ptr);
    *channels = png_get_channels(png_ptr, info_ptr);

    png_byte color_type = png_get_color_type(png_ptr, info_ptr);
    png_byte bit_depth  = png_get_bit_depth(png_ptr, info_ptr);
    if (color_type == PNG_COLOR_TYPE_PALETTE)
        png_set_palette_to_rgb(png_ptr);
    if (color_type == PNG_COLOR_TYPE_GRAY && bit_depth < 8)
        png_set_expand_gray_1_2_4_to_8(png_ptr);
    if (png_get_valid(png_ptr, info_ptr, PNG_INFO_tRNS))
        png_set_tRNS_to_alpha(png_ptr);
    if (bit_depth == 16)
        png_set_strip_16(png_ptr);
    if (color_type == PNG_COLOR_TYPE_GRAY_ALPHA || color_type == PNG_COLOR_TYPE_GRAY)
        png_set_gray_to_rgb(png_ptr);

    png_read_update_info(png_ptr, info_ptr);
    *channels = png_get_channels(png_ptr, info_ptr);

    size_t rowbytes = (*width) * (*channels) * sizeof(unsigned char);
    unsigned char* data = (unsigned char*)malloc((*height) * rowbytes);
    if (!data) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        fclose(fp);
        return NULL;
    }

    png_bytep* row_ptrs = (png_bytep*)malloc((*height) * sizeof(png_bytep));
    if (!row_ptrs) {
        free(data);
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        fclose(fp);
        return NULL;
    }
    for (int y = 0; y < *height; y++)
        row_ptrs[y] = data + y * rowbytes;

    png_read_image(png_ptr, row_ptrs);

    free(row_ptrs);
    png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
    fclose(fp);
    return data;
}

// ------------------ libpng Output ------------------
int writePNG(const char* filename, unsigned char* data, int width, int height, int channels) {
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Error opening %s for writing\n", filename);
        return -1;
    }
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) {
        fclose(fp);
        return -1;
    }
    png_infop info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_write_struct(&png_ptr, NULL);
        fclose(fp);
        return -1;
    }
    if (setjmp(png_jmpbuf(png_ptr))) {
        png_destroy_write_struct(&png_ptr, &info_ptr);
        fclose(fp);
        return -1;
    }

    png_init_io(png_ptr, fp);
    int color_type = (channels == 1 ? PNG_COLOR_TYPE_GRAY : (channels == 3 ? PNG_COLOR_TYPE_RGB : PNG_COLOR_TYPE_RGBA));
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, color_type,
                 PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_write_info(png_ptr, info_ptr);

    size_t rowbytes = width * channels * sizeof(unsigned char);
    png_bytep* row_ptrs = (png_bytep*)malloc(height * sizeof(png_bytep));
    for (int y = 0; y < height; y++)
        row_ptrs[y] = data + y * rowbytes;

    png_write_image(png_ptr, row_ptrs);
    png_write_end(png_ptr, info_ptr);

    free(row_ptrs);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
    return 0;
}

// ------------------ CUDA Kernels ------------------
// RGB->Grayscale (NCHW layout)
__global__ void grayscale_cuda(const float *rgb, float *gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    int plane = width * height;
    gray[idx] = 0.299f * rgb[0*plane + idx]
               + 0.587f * rgb[1*plane + idx]
               + 0.114f * rgb[2*plane + idx];
}

// Gaussian smoothing kernel
__global__ void gaussian_smooth_cuda(const float *in, float *out,
                                     const float *kernel, int ksize,
                                     int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int half = ksize / 2;
    float sum = 0.0f, ks = 0.0f;
    for (int dy = -half; dy <= half; dy++) {
        for (int dx = -half; dx <= half; dx++) {
            int ix = min(max(x + dx, 0), width - 1);
            int iy = min(max(y + dy, 0), height - 1);
            float val = in[iy*width + ix];
            float kval = kernel[(dy+half)*ksize + (dx+half)];
            sum += val * kval;
            ks  += kval;
        }
    }
    out[y*width + x] = (ks > 0 ? sum/ks : sum);
}

// ------------------ Host Wrappers ------------------
void create_gaussian_kernel(float *kernel, int ksize, float sigma) {
    int half = ksize / 2;
    float sum = 0.0f;
    for (int y = -half; y <= half; y++) {
        for (int x = -half; x <= half; x++) {
            float val = expf(-(x*x + y*y)/(2.0f*sigma*sigma));
            kernel[(y+half)*ksize + (x+half)] = val;
            sum += val;
        }
    }
    for (int i = 0; i < ksize*ksize; i++)
        kernel[i] /= sum;
}

void apply_gaussian_smooth(float *d_in, float *d_out, int width, int height,
                           int ksize, float sigma) {
    float *h_kernel = (float*)malloc(ksize*ksize*sizeof(float));
    create_gaussian_kernel(h_kernel, ksize, sigma);
    float *d_kernel;
    hipMalloc(&d_kernel, ksize*ksize*sizeof(float));
    hipMemcpy(d_kernel, h_kernel, ksize*ksize*sizeof(float), hipMemcpyHostToDevice);

    dim3 block(16,16), grid((width+15)/16,(height+15)/16);
    gaussian_smooth_cuda<<<grid,block>>>(d_in, d_out, d_kernel, ksize, width, height);
    hipDeviceSynchronize();

    hipFree(d_kernel);
    free(h_kernel);
}

// ------------------ Main Pipeline ------------------
int main(int argc, char* argv[]) {
    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    if (size != 4) {
        if (rank==0) fprintf(stderr, "Error: This program requires 4 MPI ranks.\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    int width=0, height=0, channels=0;
    unsigned char *img = NULL;
    if (rank == 0) {
        img = readPNG(argv[1], &width, &height, &channels);
        if (!img) MPI_Abort(MPI_COMM_WORLD, 1);
    }
    MPI_Bcast(&width,    1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&height,   1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Bcast(&channels, 1, MPI_INT, 0, MPI_COMM_WORLD);

    // Compute split offsets
    int base = height/size, rem = height%size;
    int *counts = (int*)malloc(size*sizeof(int));
    int *displs = (int*)malloc(size*sizeof(int));
    for (int i=0; i<size; i++) {
        int h = base + (i<rem);
        counts[i] = h * width * channels;
        displs[i] = (i==0?0:displs[i-1] + counts[i-1]);
    }

    int myCount = counts[rank];
    unsigned char *chunk = (unsigned char*)malloc(myCount);
    MPI_Scatterv(img, counts, displs, MPI_UNSIGNED_CHAR,
                 chunk, myCount, MPI_UNSIGNED_CHAR, 0, MPI_COMM_WORLD);

    // Convert chunk to NCHW float
    int myHeight = counts[rank]/(width*channels);
    size_t rgbSize  = 3 * width * myHeight * sizeof(float);
    size_t graySize =     width * myHeight * sizeof(float);
    float *rgbf = (float*)malloc(rgbSize);
    for(int c=0; c<3; c++)
    for(int y=0; y<myHeight; y++)
    for(int x=0; x<width; x++) {
        int pid = (y*width + x)*channels + c;
        rgbf[c*width*myHeight + y*width + x] = (float)chunk[pid];
    }

    // Launch on GPU
    hipSetDevice(rank);
    float *d_rgb, *d_gray, *d_smooth;
    hipMalloc(&d_rgb,   rgbSize);
    hipMalloc(&d_gray,  graySize);
    hipMalloc(&d_smooth,graySize);
    hipMemcpy(d_rgb, rgbf, rgbSize, hipMemcpyHostToDevice);

    dim3 block(16,16), grid((width+15)/16,(myHeight+15)/16);
    grayscale_cuda<<<grid,block>>>(d_rgb, d_gray, width, myHeight);
    hipDeviceSynchronize();

    apply_gaussian_smooth(d_gray, d_smooth, width, myHeight, 5, 1.0f);

    hipMemcpy(rgbf, d_smooth, graySize, hipMemcpyDeviceToHost);

    // Convert to uchar
    unsigned char *outc = (unsigned char*)malloc(graySize);
    for (int i=0; i<width*myHeight; i++) {
        float v = rgbf[i];
        outc[i] = (unsigned char)(v<0?0:(v>255?255:v));
    }

            // Gather single-channel results back to rank 0
    unsigned char *full = NULL;
    if (rank == 0) {
        full = (unsigned char*)malloc(width * height * sizeof(unsigned char));
    }

    int *recvCounts = (int*)malloc(size * sizeof(int));
    int *recvDispls = (int*)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        int h_i = counts[i] / (width * channels);
        recvCounts[i] = h_i * width;
        int offsetRows = displs[i] / (width * channels);
        recvDispls[i] = offsetRows * width;
    }
    int sendCount = myHeight * width;

    MPI_Gatherv(outc, sendCount, MPI_UNSIGNED_CHAR,
                full, recvCounts, recvDispls,
                MPI_UNSIGNED_CHAR, 0, MPI_COMM_WORLD);

    if (rank == 0) {
        writePNG("output.png", full, width, height, 1);
        free(full);
    }

    // Cleanup
    free(recvCounts);
    free(recvDispls);
    free(img);
    free(chunk);
    free(rgbf);
    free(outc);
    free(counts);
    free(displs);
    hipFree(d_rgb);
    hipFree(d_gray);
    hipFree(d_smooth);
    MPI_Finalize();
    return 0;
}